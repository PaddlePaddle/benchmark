#include "hip/hip_runtime.h"

#include "stdio.h"

int main() {
  int version = -1;
  hipError_t err = hipRuntimeGetVersion(&version); 
  if (err != hipSuccess) {
    printf("%s\n", hipGetErrorString(err));
    return -1;
  }
  int major = version / 1000;
  int minor = (version % 100) / 10; 
  printf("%d.%d\n", major, minor);
  return 0;
}
